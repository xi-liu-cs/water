
#include <hip/hip_runtime.h>
__global__ void mainImage(uchar4 * fragColor, float iTime)
{
	int width = 1024,
	height = 1024;
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x,
	y = blockIdx.y * blockDim.y + threadIdx.y,
	i = x + width * y;
	float2 iResolution = make_float2(width, height),
	fragCoord = make_float2(x, y),
	uv = make_float2(fragCoord.x / iResolution.x, fragCoord.y / iResolution.y);
	float4 color = make_float4(uv.x, uv.y, 0.0, 1.0);
	fragColor[i] = make_uchar4(color.x * 255, color.y * 255, color.z * 255, 255);
}