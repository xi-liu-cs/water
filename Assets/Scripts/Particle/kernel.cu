
#include <hip/hip_runtime.h>
__global__ void mainImage(uchar4 * fragColor, float iTime)
{
	int width = 1024;
	int height = 1024;
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int i = x + width * y;
	float2 iResolution = make_float2((float)width, (float)height);
	float2 fragCoord = make_float2((float)x, (float)y);
	float2 uv = make_float2(fragCoord.x / iResolution.x, fragCoord.y / iResolution.y);
	float4 color = make_float4(uv.x, uv.y, 0.0, 1.0);
	fragColor[i] = make_uchar4(color.x * 255, color.y * 255, color.z * 255, 255);
}